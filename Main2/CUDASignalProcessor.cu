#include "CUDASignalProcessor.h"

CUDASignalProcessor::CUDASignalProcessor(size_t datalen) : SignalProcessor(datalen)
{
	hipError_t status;

	this->err = 0;
	this->time = 0;

	status = hipSetDevice(0);
	if (check_cuda_status(status)) goto fin;

	status = hipMalloc((void**) &(this->re), datalen * sizeof(float));
	if (check_cuda_status(status)) goto fin;

	status = hipMalloc((void**) &(this->im), datalen * sizeof(float));
	if (check_cuda_status(status)) goto fin;

fin:
}

CUDASignalProcessor::~CUDASignalProcessor()
{
	hipFree(this->re);
	hipFree(this->im);
}

int CUDASignalProcessor::check_cuda_status(hipError_t status)
{
	if (status != hipSuccess)
	{
		this->err = 1;
		std::cout << "cuda err: " << hipGetErrorString(status) << std::endl;
	}
	return this->err;
}

void CUDASignalProcessor::process_buffer(float* real, float* imaginary, size_t readcount)
{
	hipError_t status;

	auto start = std::chrono::high_resolution_clock::now();

	status = hipMemcpy(this->re, real, readcount * sizeof(float), hipMemcpyHostToDevice);
	if (check_cuda_status(status)) return;

	status = hipMemcpy(this->im, imaginary, readcount * sizeof(float), hipMemcpyHostToDevice);
	if (check_cuda_status(status)) return;

	this->exec_kernel(this->re, this->im, readcount);

	status = hipGetLastError();
	if (check_cuda_status(status)) return;

	status = hipDeviceSynchronize();
	if (check_cuda_status(status)) return;

	status = hipMemcpy(real, this->re, readcount * sizeof(float), hipMemcpyDeviceToHost);
	if (check_cuda_status(status)) return;

	status = hipMemcpy(imaginary, this->im, readcount * sizeof(float), hipMemcpyDeviceToHost);
	if (check_cuda_status(status)) return;

	auto end = std::chrono::high_resolution_clock::now();
	
	this->time += std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

}

void CUDASignalProcessor::reset()
{
	this->time = 0;
}

int CUDASignalProcessor::get_err()
{
	return this->err;
}

long long int CUDASignalProcessor::get_time()
{
	return this->time;
}